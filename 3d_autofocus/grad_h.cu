#include "hip/hip_runtime.h"
#include "grad_h.h"

#if MATLAB_MEX_FILE
#include "mex.h"
#define PRINTF mexPrintf
#else
#define PRINTF printf
#endif

#include <cmath>
#include <vector>
#include <thread>
#include <assert.h>

#include <hip/hip_runtime.h>

using namespace std;

__global__ void reduce0(double *P, double *z, double *Br, double *Bi, int K, bool is_z_r) {
  extern __shared__ double sdata[];

  double *p_r = sdata;

  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (is_z_r) {
    p_r[tid] = Br[i] * cos(P[i]) + Bi[i] * sin(P[i]);
  }
  else {
    p_r[tid] = Bi[i] * cos(P[i]) - Br[i] * sin(P[i]);
  }

  __syncthreads();

  // do reduction in shared mem
  for(unsigned int s = 1; s < blockDim.x; s *= 2) {
    if (tid % (2*s) == 0) {
      p_r[tid] += p_r[tid + s];
    }

    __syncthreads();
  }

  // write result for this block to global mem
  if (tid == 0) {
    z[blockIdx.x] = p_r[0];
  }
}

// TODO: Nice doc comments
void gradH(double *phi_offsets, const double *Br, const double *Bi,
        double *grad, size_t K, size_t B_len)
{
    vector<double> P(phi_offsets, phi_offsets + K);

    PRINTF("In gradH, about to compute Z\n");
    PRINTF("Computed Z\n");
    double H_not = H(P, Br, Bi, K, B_len);
    PRINTF("Computed H_not\n");

    auto Pr_k(P.begin());

    while (Pr_k != P.end()) {
      if (Pr_k != P.begin()) {
        *(Pr_k - 1) -= delta;
      }

      *Pr_k++ += delta;

      populate_grad_k(grad++, H_not, P, Br, Bi, K, B_len);
    }
}

#define funcCheck(stmt) {                                            \
  hipError_t err = stmt;                                          \
  if (err != hipSuccess)                                          \
  {                                                                \
    printf( "Failed to run stmt %d\n", __LINE__);                 \
    printf( "Got CUDA error ...  %s\n", hipGetErrorString(err)); \
    return -1;                                                   \
  }                                                                \
}

// Returns the entropy of the complex image `Z`
double H(const vector<double> P, const double *Br, const double *Bi,
        size_t K, size_t B_len)
{
    size_t N = B_len / K;
    double Ez = 0, entropy = 0;

    assert(B_len % K == 0); // length(B) should always be a multiple of K

    double *Z_mag = new double[N];

    // ------------------------------------------------------------------------
    // Form Z_mag
    // ------------------------------------------------------------------------
    const int bs = 4; // NOTE: Must be a multiple of K
    const int nT = K / bs;

    double *z_r = (double *)malloc(bs * sizeof(double));
    double *z_i = (double *)malloc(bs * sizeof(double));

    double *d_P, *d_Br, *d_Bi;
    double *d_z_r = NULL, *d_z_i = NULL;

    funcCheck(hipMalloc((void **)&d_P,  K * sizeof(double)));
    funcCheck(hipMalloc((void **)&d_Br, B_len * sizeof(double)));
    funcCheck(hipMalloc((void **)&d_Bi, B_len * sizeof(double)));

    funcCheck(hipMalloc((void **)&d_z_r, bs * sizeof(double)));
    funcCheck(hipMalloc((void **)&d_z_i, bs * sizeof(double)));

    funcCheck(hipMemcpy(d_P, &P[0], K * sizeof(double), hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(d_Br, Br, B_len * sizeof(double), hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(d_Bi, Bi, B_len * sizeof(double), hipMemcpyHostToDevice));

    for (size_t n = 0; n < N; ++n) {
      reduce0<<<bs, nT, nT * sizeof(double)>>>(d_P, d_z_r, d_Br + n * K, d_Bi + n * K, K, true);
      reduce0<<<bs, nT, nT * sizeof(double)>>>(d_P, d_z_i, d_Br + n * K, d_Bi + n * K, K, false);

      funcCheck(hipMemcpy(z_r, d_z_r, bs * sizeof(double), hipMemcpyDeviceToHost));
      funcCheck(hipMemcpy(z_i, d_z_i, bs * sizeof(double), hipMemcpyDeviceToHost));

      for (size_t b(1); b < bs; ++b) { z_r[0] += z_r[b]; }
      for (size_t b(1); b < bs; ++b) { z_i[0] += z_i[b]; }

      Z_mag[n] = *z_r * *z_r + *z_i * *z_i;
    }

    // Returns the total image energy of the complex image Z_mag given the
    // magnitude of // the pixels in Z_mag
    for (size_t n = 0; n < N; ++n) { Ez += Z_mag[n]; }

    funcCheck(hipFree(d_P));
    funcCheck(hipFree(d_Br));
    funcCheck(hipFree(d_Bi));

    funcCheck(hipFree(d_z_r));
    funcCheck(hipFree(d_z_i));

    free(z_r);
    free(z_i);

    double z_intensity = 0;
    for (size_t n = 0; n < N; ++n) {
        z_intensity = Z_mag[n] / Ez;
        entropy += z_intensity * log(z_intensity);
    }

    delete[] Z_mag;
    return - entropy;
}

void populate_grad_k(double *grad_i, double H_not, const vector<double> P,
        const double *Br, const double *Bi, size_t K,
        size_t B_len)
{
    double H_i = H(P, Br, Bi, K, B_len);
    *grad_i = (H_i - H_not) / delta;
}

