#include "hip/hip_runtime.h"
#include "grad_h.h"

#if MATLAB_MEX_FILE
#include "mex.h"
#define PRINTF mexPrintf
#else
#define PRINTF printf
#endif

#include <cmath>
#include <vector>
#include <thread>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

using namespace std;

const int nT = 1024;

#define checkCuda(result) checkCudaInner(result, __LINE__)

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCudaInner(hipError_t result, int lineno)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s on line %d\n", hipGetErrorString(result), lineno);
    assert(result == hipSuccess);
  }
  return result;
}

__device__ inline double entropy(double acc, double Ez)
{
  return (acc - Ez * log(Ez)) / Ez;
}

// B is a matrix with K columns and N rows
template<typename T>
__global__ void computeGrad(const T * __restrict__ Br, const T * __restrict__ Bi, 
    const T * __restrict__ Zr, const T * __restrict__ Zi,
    const T * __restrict__ Ar, const T * __restrict__ Ai,
    T * __restrict__ grad, size_t N, size_t K, double H0, double delta)
{
  extern __shared__ T sdata[];

  T *Ez = sdata, *acc = &sdata[blockDim.x];

  T x(0.0), y(0.0);

  const T * Br_col = &Br[blockIdx.x];
  const T * Bi_col = &Bi[blockIdx.x];

  // Accumulate per thread partial sum
  for (int i = threadIdx.x; i < N; i += blockDim.x) {
    int col_idx = i * K;

    double Zn_r = Ar[blockIdx.x] * Br_col[col_idx] - Ai[blockIdx.x] * Bi_col[col_idx] + Zr[i];
    double Zn_i = Ar[blockIdx.x] * Bi_col[col_idx] + Ai[blockIdx.x] * Br_col[col_idx] + Zi[i];

    double Zn_mag = Zn_r * Zn_r + Zn_i * Zn_i;

    x += Zn_mag;
    y += Zn_mag * log(Zn_mag);
  }

  // load thread partial sum into shared memory
  Ez[threadIdx.x] = x;
  acc[threadIdx.x] = y;

  __syncthreads();

  for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
    if (threadIdx.x < offset) {
      Ez[threadIdx.x] += Ez[threadIdx.x + offset];
      acc[threadIdx.x] += acc[threadIdx.x + offset];
    }
    __syncthreads();
  }

  // thread 0 writes the final result
  if (threadIdx.x == 0) {
    grad[blockIdx.x] = (-entropy(acc[0], Ez[0]) - H0) / delta;
  }
}

// B is a matrix with K columns and N rows
template<typename T>
__global__ void kernelSum(const T * __restrict__ Br, const T * __restrict__ Bi, 
    T * __restrict__ Zr, T * __restrict__ Zi,
    const size_t K, T * __restrict__ Z_mag, const T * __restrict__ P)
{
  extern __shared__ T sdata[];

  T *s1 = sdata, *s2 = &sdata[blockDim.x];

  T x(0.0), y(0.0);

  const T * Br_row = &Br[blockIdx.x * K];
  const T * Bi_row = &Bi[blockIdx.x * K];

  // Accumulate per thread partial sum
  double sin, cos;
  for (int i = threadIdx.x; i < K; i += blockDim.x) {
    sincos(P[i % K], &sin, &cos);
    x += Br_row[i] * cos + Bi_row[i] * sin;
    y += Bi_row[i] * cos - Br_row[i] * sin;
  }

  // load thread partial sum into shared memory
  s1[threadIdx.x] = x;
  s2[threadIdx.x] = y;

  __syncthreads();

  for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
    if (threadIdx.x < offset) {
      s1[threadIdx.x] += s1[threadIdx.x + offset];
      s2[threadIdx.x] += s2[threadIdx.x + offset];
    }
    __syncthreads();
  }

  // thread 0 writes the final result
  if (threadIdx.x == 0) {
    Zr[blockIdx.x] = s1[0];
    Zi[blockIdx.x] = s2[0];

    Z_mag[blockIdx.x] = s1[0] * s1[0] + s2[0] * s2[0];
  }
}

template <class T>
__global__ void computeEntropy(T *g_idata, T *g_odata, double *Ez, unsigned int n)
{
  extern __shared__ T sdata[];

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  sdata[tid] = (i < n) ? (g_idata[i] / *Ez) * log(g_idata[i] / *Ez) : 0;

  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s=1; s < blockDim.x; s *= 2)
  {
    // modulo arithmetic is slow!
    if ((tid % (2*s)) == 0)
    {
      sdata[tid] += sdata[tid + s];
    }

    __syncthreads();
  }

  // write result for this block to global mem
  if (tid == 0) {
    g_odata[blockIdx.x] = sdata[0];
  }
}

__global__ void computeAlpha(const double *P, double sin_delt, double cos_delt,
    double *Ar, double *Ai, size_t K)
{
  unsigned int k = blockIdx.x*blockDim.x + threadIdx.x;

  double sin_phi, cos_phi;
  if (k < K) {
    sincos(P[k], &sin_phi, &cos_phi);

    Ar[k] = (-sin_delt) * sin_phi + cos_delt * cos_phi - cos_phi;
    Ai[k] = sin_delt * (-cos_phi) - cos_delt * sin_phi + sin_phi;
  }
}

// Returns the entropy of the complex image `Z`
double H_not(const double *d_P, double *d_Br, double *d_Bi, double *Zr, double
    *Zi, size_t K, size_t B_len, hipStream_t *stream, int nStreams, int
    streamSize)
{
  size_t N = B_len / K;
  double entropy(0);

  assert(B_len % K == 0); // length(B) should always be a multiple of K

  double *d_Z_mag = NULL;

  checkCuda(hipMalloc((void **)&d_Z_mag, N * sizeof(double)));

  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    int Z_offset = i * (N / nStreams);

    kernelSum<double><<<N / nStreams, nT, 2 * nT * sizeof(double),
      stream[i]>>>(&d_Br[offset], &d_Bi[offset], &Zr[Z_offset], &Zi[Z_offset], K, &d_Z_mag[Z_offset], d_P);
  }

  hipDeviceSynchronize();

  int bs = (N + nT - 1) / nT; // cheap ceil()

  double *d_Ez = NULL;
  double *d_accum = NULL;
  double *accum = NULL;

  checkCuda(hipHostMalloc((void **)&accum, bs * sizeof(double)));
  checkCuda(hipMalloc((void **)&d_accum, bs * sizeof(double)));
  checkCuda(hipMalloc((void **)&d_Ez, sizeof(double)));

  hipblasHandle_t handle;

  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

  // hipblasDasum sums the absolute values, but Z_mag is always positive so this
  // is correct
  hipblasDasum(handle, N, d_Z_mag, 1, d_Ez);

  computeEntropy<double><<<bs, nT, nT * sizeof(double)>>>(d_Z_mag, d_accum, d_Ez, N);

  checkCuda(hipMemcpy(accum, d_accum, bs * sizeof(double), hipMemcpyDeviceToHost));

  for (size_t b(0); b < bs; ++b) { entropy += accum[b]; }

  checkCuda(hipFree(d_Z_mag));
  checkCuda(hipFree(d_accum));
  checkCuda(hipFree(d_Ez));
  hipblasDestroy(handle);

  return -entropy;
}

void gradH_priv(double *d_P, const double *d_Br, const double *d_Bi,
    double *d_grad, const double *d_Zr, const double *d_Zi, size_t K, size_t B_len, double H0)
{
  const size_t N = B_len / K;

  double *d_Ar, *d_Ai;

  checkCuda(hipMalloc((void **)&d_Ar,  K * sizeof(double)));
  checkCuda(hipMalloc((void **)&d_Ai,  K * sizeof(double)));

  double sin_delt, cos_delt;

  sincos(delta, &sin_delt, &cos_delt);

  int bs = (K + nT - 1) / nT; // cheap ceil
  computeAlpha<<<bs, nT, 0>>>(d_P, sin_delt, cos_delt, d_Ar, d_Ai, K);

  computeGrad<double><<<K, nT, 2 * nT * sizeof(double)>>>(d_Br, d_Bi, d_Zr, d_Zi, d_Ar, d_Ai, d_grad, N, K, H0, delta);

  checkCuda(hipFree(d_Ai));
  checkCuda(hipFree(d_Ar));
}

// TODO: Nice doc comments
void gradH(double *phi_offsets, const double *Br, const double *Bi,
    double *grad, size_t K, size_t B_len)
{
  // const int maxMem = 2147483648; // 2 GB
  const int N = B_len / K;

  // Solve for K:
  // (4 * K + 2 * N + 2 * K * N) * sizeof(double);
  // int K_prime = min(((maxMem / sizeof(double)) - 2 * N) / (4 + 2 * N), K);

  const int nStreams = (B_len > 8) ? 8 : 1;
  const int streamSize = B_len / nStreams;

  hipStream_t stream[nStreams];

  for (int i = 0; i < nStreams; ++i)
    checkCuda(hipStreamCreate(&stream[i]));

  double *d_Br, *d_Bi, *d_P, *d_Zr, *d_Zi, *d_grad;

  // TODO: Use pinned memory
  checkCuda(hipMalloc((void **)&d_P,  K * sizeof(double)));

  checkCuda(hipMemcpy(d_P, phi_offsets, K * sizeof(double), hipMemcpyHostToDevice));

  // TODO: Use pinned memory
  checkCuda(hipMalloc((void **)&d_Br, B_len * sizeof(double)));
  checkCuda(hipMalloc((void **)&d_Bi, B_len * sizeof(double)));

  checkCuda(hipMalloc((void **)&d_Zr, N * sizeof(double)));
  checkCuda(hipMalloc((void **)&d_Zi, N * sizeof(double)));

  checkCuda(hipMalloc((void **)&d_grad, K * sizeof(double)));

  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;

    checkCuda(hipMemcpyAsync(&d_Br[offset], &Br[offset], streamSize * sizeof(double), hipMemcpyHostToDevice, stream[i]));
    checkCuda(hipMemcpyAsync(&d_Bi[offset], &Bi[offset], streamSize * sizeof(double), hipMemcpyHostToDevice, stream[i]));
  }

  PRINTF("In gradH, about to compute Z\n");
  PRINTF("Computed Z\n");
  double H0 = H_not(d_P, d_Br, d_Bi, d_Zr, d_Zi, K, B_len, stream, nStreams, streamSize);
  PRINTF("Computed H_not\n");

  gradH_priv(d_P, d_Br, d_Bi, d_grad, d_Zr, d_Zi, K, B_len, H0);

  checkCuda(hipMemcpy(grad, d_grad, K * sizeof(double), hipMemcpyDeviceToHost));

  checkCuda(hipFree(d_Br));
  checkCuda(hipFree(d_Bi));

  checkCuda(hipFree(d_Zr));
  checkCuda(hipFree(d_Zi));

  checkCuda(hipFree(d_P));

  for (int i = 0; i < nStreams; ++i)
    checkCuda(hipStreamDestroy(stream[i]));
}
