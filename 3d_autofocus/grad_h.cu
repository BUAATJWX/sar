#include "hip/hip_runtime.h"
#include "grad_h.h"

#if MATLAB_MEX_FILE
#include "mex.h"
#define PRINTF mexPrintf
#else
#define PRINTF printf
#endif

#include <cmath>
#include <vector>
#include <thread>
#include <assert.h>

#include <hip/hip_runtime.h>

using namespace std;

__global__ void reduce0(double *P, double *Br_shift, double *Bi_shift, double *Br, double *Bi, size_t K, size_t B_len)
{
  unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;

  if (n < B_len) {
    Br_shift[n] = Br[n] * cos(P[n % K]) + Bi[n] * sin(P[n % K]);
    Bi_shift[n] = Bi[n] * cos(P[n % K]) - Br[n] * sin(P[n % K]);
  }
}

__global__ void reduce1(double *Z_mag, double *z_r, double *z_i, size_t N)
{
  unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;

  if (n < N) {
    Z_mag[n] = z_r[n] * z_r[n] + z_i[n] * z_i[n];
  }
}

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};
template<>
struct SharedMemory<double>
{
    __device__ inline operator       double *()
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};

template <class T>
__global__ void sum(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=1; s < blockDim.x; s *= 2)
    {
        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0)
        {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <class T>
__global__ void reduce2(T *g_idata, T *g_odata, double Ez, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? (g_idata[i] / Ez) * log(g_idata[i] / Ez) : 0;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=1; s < blockDim.x; s *= 2)
    {
        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0)
        {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// TODO: Nice doc comments
void gradH(double *phi_offsets, const double *Br, const double *Bi,
        double *grad, size_t K, size_t B_len)
{
    vector<double> P(phi_offsets, phi_offsets + K);

    PRINTF("In gradH, about to compute Z\n");
    PRINTF("Computed Z\n");
    double H_not = H(P, Br, Bi, K, B_len);
    PRINTF("Computed H_not\n");

    auto Pr_k(P.begin());

    while (Pr_k != P.end()) {
      if (Pr_k != P.begin()) {
        *(Pr_k - 1) -= delta;
      }

      *Pr_k++ += delta;

      populate_grad_k(grad++, H_not, P, Br, Bi, K, B_len);
    }
}

#define funcCheck(stmt) {                                            \
  hipError_t err = stmt;                                          \
  if (err != hipSuccess)                                          \
  {                                                                \
    printf( "Failed to run stmt %d\n", __LINE__);                 \
    printf( "Got CUDA error ...  %s\n", hipGetErrorString(err)); \
    return -1;                                                   \
  }                                                                \
}

// Returns the entropy of the complex image `Z`
double H(const vector<double> P, const double *Br, const double *Bi,
        size_t K, size_t B_len)
{
    size_t N = B_len / K;
    double Ez(0), entropy(0);

    assert(B_len % K == 0); // length(B) should always be a multiple of K

    // ------------------------------------------------------------------------
    // Form Z_mag
    // ------------------------------------------------------------------------
    double *Z_mag    = NULL;
    double *z_r      = NULL;
    double *z_i      = NULL;
    double *Br_shift = NULL;
    double *Bi_shift = NULL;

    funcCheck(hipHostMalloc((void **)&Z_mag, N * sizeof(double)));
    funcCheck(hipHostMalloc((void **)&z_r, N * sizeof(double)));
    funcCheck(hipHostMalloc((void **)&z_i, N * sizeof(double)));
    funcCheck(hipHostMalloc((void **)&Br_shift, B_len * sizeof(double)));
    funcCheck(hipHostMalloc((void **)&Bi_shift, B_len * sizeof(double)));

    const int nT = 1024;
    int bs = (B_len + nT - 1) / nT; // cheap ceil()

    double *d_P, *d_Br, *d_Bi;
    double *d_Br_shift = NULL, *d_Bi_shift = NULL;
    double *d_z_r = NULL, *d_z_i = NULL;
    double *d_Z_mag = NULL;

    funcCheck(hipMalloc((void **)&d_P,  K * sizeof(double)));
    funcCheck(hipMalloc((void **)&d_Br, B_len * sizeof(double)));
    funcCheck(hipMalloc((void **)&d_Bi, B_len * sizeof(double)));

    funcCheck(hipMalloc((void **)&d_Br_shift, B_len * sizeof(double)));
    funcCheck(hipMalloc((void **)&d_Bi_shift, B_len * sizeof(double)));

    funcCheck(hipMalloc((void **)&d_z_r, N * sizeof(double)));
    funcCheck(hipMalloc((void **)&d_z_i, N * sizeof(double)));
    funcCheck(hipMalloc((void **)&d_Z_mag, N * sizeof(double)));

    funcCheck(hipMemcpy(d_P, &P[0], K * sizeof(double), hipMemcpyHostToDevice));

    funcCheck(hipMemcpy(d_Br, Br, B_len * sizeof(double), hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(d_Bi, Bi, B_len * sizeof(double), hipMemcpyHostToDevice));

    reduce0<<<bs, nT, 0>>>(d_P, d_Br_shift, d_Bi_shift, d_Br, d_Bi, K, B_len);

    funcCheck(hipMemcpy(Br_shift, d_Br_shift, B_len * sizeof(double), hipMemcpyDeviceToHost));
    funcCheck(hipMemcpy(Bi_shift, d_Bi_shift, B_len * sizeof(double), hipMemcpyDeviceToHost));

    // TODO: Explore reduction kernel
    for (size_t n = 0; n < B_len; ++n) {
      z_r[n / K] += Br_shift[n];
      z_i[n / K] += Bi_shift[n];
    }

    funcCheck(hipMemcpy(d_z_r, z_r, N * sizeof(double), hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(d_z_i, z_i, N * sizeof(double), hipMemcpyHostToDevice));

    reduce1<<<bs, nT>>>(d_Z_mag, d_z_r, d_z_i, N);

    funcCheck(hipFree(d_P));
    funcCheck(hipFree(d_Br));
    funcCheck(hipFree(d_Bi));

    funcCheck(hipFree(d_Br_shift));
    funcCheck(hipFree(d_Bi_shift));

    funcCheck(hipFree(d_z_r));
    funcCheck(hipFree(d_z_i));

    // Returns the total image energy of the complex image Z_mag given the
    // magnitude of // the pixels in Z_mag
    // TODO: Explore reduction kernel
    bs = (N + nT - 1) / nT; // cheap ceil()

    double *accum   = NULL;
    double *d_accum = NULL;

    funcCheck(hipHostMalloc((void **)&accum, bs * sizeof(double)));
    funcCheck(hipMalloc((void **)&d_accum, bs * sizeof(double)));

    sum<double><<<bs, nT, nT * sizeof(double)>>>(d_Z_mag, d_accum, N);
    funcCheck(hipMemcpy(accum, d_accum, bs * sizeof(double), hipMemcpyDeviceToHost));
    for (size_t b(0); b < bs; ++b) { Ez += accum[b]; }


    reduce2<double><<<bs, nT, nT * sizeof(double)>>>(d_Z_mag, d_accum, Ez, N);
    funcCheck(hipMemcpy(accum, d_accum, bs * sizeof(double), hipMemcpyDeviceToHost));
    for (size_t b(0); b < bs; ++b) { entropy += accum[b]; }

    funcCheck(hipFree(d_Z_mag));
    funcCheck(hipFree(d_accum));

    funcCheck(hipHostFree(accum));
    funcCheck(hipHostFree(Z_mag));
    funcCheck(hipHostFree(z_r));
    funcCheck(hipHostFree(z_i));
    funcCheck(hipHostFree(Br_shift));
    funcCheck(hipHostFree(Bi_shift));

    return - entropy;
}

void populate_grad_k(double *grad_i, double H_not, const vector<double> P,
        const double *Br, const double *Bi, size_t K,
        size_t B_len)
{
    double H_i = H(P, Br, Bi, K, B_len);
    *grad_i = (H_i - H_not) / delta;
}

